// Genetic Algorithm on Multiple GPUs


// ga_gpu.cu contains all the CUDA routines




#include<hip/hip_runtime.h>

#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>


// Extern variables to make them available here in the cuda file

// Result from last compute of world.
extern unsigned char *currentGen;
// Current state of world. 
extern unsigned char *nextGen;
// Map
extern double *map;
// Fitness
extern double *fitness;
// Global roulette indices
extern unsigned long long *globalRouletteWheel;

// Problem Sizes
// Population Size
// Cities/Genome Length
extern unsigned long long popSize;
extern unsigned long long cities;
extern unsigned long long globalSize;



// Bind GPUs to the ranks
extern "C" void bindGPUs(int rank)
{

  // Bind GPUs to the ranks
  int cudaDeviceCount;
  int cE;

  if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
  {
    printf(" Unable to determine cuda device count, error is %d, count is %d\n",
      cE, cudaDeviceCount );
    exit(-1);
  }

  if( (cE = hipSetDevice( rank % cudaDeviceCount )) != hipSuccess )
  {
    printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
      rank, (rank % cudaDeviceCount), cE);
    exit(-1);
  }



}



// Initialization Routine
extern "C" void popAlloc(int cities, int popSize){

  // Allocate the memory
  globalSize = cities*popSize;
  hipMallocManaged(&currentGen, globalSize*(sizeof(unsigned char)));
  hipMallocManaged(&nextGen, globalSize*(sizeof(unsigned char)));



}


extern "C" void mapAlloc(unsigned long long cities){
// cuda malloc manage the parsed map ----> copy it into variable called map
  unsigned long long mapsize = cities*cities;
  hipMallocManaged(&map, mapsize*(sizeof(double)));
}



extern "C" void fitnessAlloc(int num_ranks, int rank, int popSize){

  // Allocate the memory
  int localSize = popSize/num_ranks;
  hipMallocManaged(&fitness, localSize*(sizeof(double)));
  



}





// Fitness calculation Kernel
__global__ void fitness_kernel(int num_ranks, int rank, int popSize, int cities, unsigned char *currentGen, double *fitness, double *map){
  
  int localPopSize = popSize/num_ranks;
 
 
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  for (;
       index < localPopSize;
       index += blockDim.x * gridDim.x){

    int j = 0;
    double sum = 0;

    int a = 0;
    int b = 0;
    double distance=0;

    for(j=0;j<cities-1;j++){
      
      
      a = currentGen[rank*cities*localPopSize + index*cities + j];
      b = currentGen[rank*cities*localPopSize + index*cities + j+1];

      distance = map[a*cities + b];

      sum = sum + distance;
    }

    // add the last two
    a = currentGen[rank*cities*localPopSize + index*cities];
    b = currentGen[rank*cities*localPopSize + index*cities + cities-1];
    distance = map[a*cities + b];
    sum = sum + distance;

    fitness[index] = sum;

  }

  

}





// Function to launch the Fitness Kernel
extern "C" bool fitness_kernelLaunch (int num_ranks, int rank, int popSize, int cities)
{
  // get the minimum number of required blockCount
  //size_t blockCount = (worldWidth * worldHeight + threadsCount - 1)/threadsCount;
  size_t threadsCount = 1024;
  size_t blockCount = popSize/threadsCount+1;
  if(blockCount == 0) blockCount++;
  //printf("blockCount: %d \n", blockCount);
  fitness_kernel<<<blockCount, threadsCount>>>(num_ranks, rank, popSize, cities, currentGen, fitness, map);
  hipDeviceSynchronize();

  return false;
}

















__global__ void parents_kernel(int num_ranks, int popSize, int cities, unsigned char *currentGen, unsigned char *nextGen, int r_wheel_length, unsigned long long *globalRouletteWheel){
  
  int localPopSize = popSize/num_ranks;
 
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  
  int nextgen_i = index + (index/r_wheel_length)*(localPopSize - r_wheel_length);
  int currentgen_i = globalRouletteWheel[index];

  int j = 0;
  for (;
       index < r_wheel_length*num_ranks;
       index += blockDim.x * gridDim.x){

    for(j=0;j<cities;j++){
      nextGen[nextgen_i*cities + j] = currentGen[currentgen_i*cities +j];
    }


  }
}
 
// Function to launch the Fitness Kernel
extern "C" bool parents_kernelLaunch (int num_ranks, int r_wheel_length)
{
  // get the minimum number of required blockCount
  //size_t blockCount = (worldWidth * worldHeight + threadsCount - 1)/threadsCount;
  size_t threadsCount = 1024;
  size_t blockCount = r_wheel_length/threadsCount+1;
  if(blockCount == 0) blockCount++;
  //printf("blockCount: %d \n", blockCount);
  parents_kernel<<<blockCount, threadsCount>>>(num_ranks, popSize, cities, currentGen, nextGen, r_wheel_length, globalRouletteWheel);
  hipDeviceSynchronize();

  return false;
}





// Function to launch the cum sum kernel
// Kernel for cum sum (scan)
void cum_kernelLaunch(){
}
//

// Function to launch the copying for next generation parents
// Kernel for copying


// Function to launch the crossover kernel
// crossover kernel






